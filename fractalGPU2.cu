#include "hip/hip_runtime.h"
//nvcc fractalGPU2.cu -o temp2 -lglut -lGL -lm

#include <GL/glut.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

unsigned int window_width = 1024;
unsigned int window_height = 1024;

float zoom = 0.035;

float xOffset = 950.0;
float yOffset = 450.0;

float a = 0.365;
float b = 0.11;

dim3 dimBlock;
dim3 dimGrid;

float *pixels_CPU;
float *pixels_GPU;

float xValue (int x) 
{
	float xVal = (2.0f * (((x * zoom)/(window_width-1)))-zoom);
	return xVal;
}

float yValue (int y) 
{
	float yVal = ((-2.0f)*(((y * zoom)/(window_height-1)))+zoom);
	return yVal;
}

__device__ float color (int i, int j) 
{
	float x, y, mag,maxMag,t1, a, b;
	float maxCount = 200;
	float count = 0;
	maxMag = 10;
	mag = 0.0;
	a = 0.365;	
	b = 0.11;
	x = xValue(i);
	y = yValue(j);
	
	while (mag < maxMag && count < maxCount) 
	{
		t1 = x;	
		x = pow(x,2) - pow(y,2) + a;
		y = (2 * t1 * y) + b;
		mag = sqrt(pow(x,2)+pow(y,2));
		count += 0.75;
	}
	return(count*1.0f/maxCount);
}

__global__ julea(pixel)
{
	k=0;
	for(int i = 0; i < window_width; i++) 
	{
		for(int j = 0; j < window_height; j++) 
		{
			pixels_CPU[k] = color(i+yOffset,j+xOffset);
			pixels_CPU[k+1] = 0.125; 
			pixels_CPU[k+2] = 0.30;
			k=k+3;
		}
	}
}

void display(void) { 
	int i,j,k;
	
	dimBlock.x = 1024;
	dimBlock.y = 1;
	dimBlock.z = 1;
	dimGrid.x = 1024;
	dimGrid.y = 1;
	dimGrid.z = 1;

	pixels_CPU = (float *)malloc(window_width*window_height*3*sizeof(float));
	hipMalloc((void**)&pixels_GPU, window_width*window_height*3*sizeof(float));
	
	julea<<<GridConfig, BlockConfig>>>(pixels_GPU);
	
	hipMemcpy( pixels_CPU, pixels_GPU, window_width*window_height*3*sizeof(float), hipMemcpyDeviceToHost );

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels_CPU); 
	glFlush(); 
}

int main(int argc, char** argv)
{ 
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	glutMainLoop();
}

