#include "hip/hip_runtime.h"
//nvcc fractalSimpleGPU.cu -o temp2 -lglut -lGL -lm

#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>


#define A  -0.52
#define B  0.57

#define xMin -2.0
#define xMax  2.0
#define yMin -2.0
#define yMax  2.0

unsigned int window_width = 1024;
unsigned int window_height = 1024;

float steps = 1024.0;
float stepSizeX = (xMax - xMin)/steps;
float stepSizeY = (yMax - yMin)/steps;

dim3 dimBlock;
dim3 dimGrid;

float *pixels_CPU;
float *pixels_GPU;

void __global__ julia(float *pixels, float stepX, float stepY)
{
	float mag,t1;
	
	float maxCount = 200;
	float count = 0;
	float maxMag = 10;
	
	float x = threadIdx.x*stepX + xMin;
	float y = blockIdx.x*stepY + yMin;
	int k = threadIdx.x*3 + blockDim.x*blockIdx.x*3;
	
	mag = 0.0;
	while (mag < maxMag && count < maxCount) 
	{
		t1 = x;	
		x = x*x - y*y + A;
		y = (2.0 * t1 * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	if(count < maxCount) pixels[k] = 0.0;
	else pixels[k] = 0.0;
	pixels[k+1] = 0.0; 
	pixels[k+2] = 0.0;
	
}

void display(void) 
{ 	
	dimBlock.x = 1024;
	dimBlock.y = 1;
	dimBlock.z = 1;
	dimGrid.x = 1024;
	dimGrid.y = 1;
	dimGrid.z = 1;

	pixels_CPU = (float *)malloc(window_width*window_height*3*sizeof(float));
	hipMalloc((void**)&pixels_GPU, window_width*window_height*3*sizeof(float));
	
	julia<<<GridConfig, BlockConfig>>>(pixels_GPU, stepSizeX, stepSizeY);
	
	hipMemcpy( pixels_CPU, pixels_GPU, window_width*window_height*3*sizeof(float), hipMemcpyDeviceToHost );

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels); 
	glFlush(); 
}

int main(int argc, char** argv)
{ 
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	glutMainLoop();
}

