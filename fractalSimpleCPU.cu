#include "hip/hip_runtime.h"
//nvcc fractalSimpleCPU.cu -o temp -lglut -lGL -lm

#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>


#define A  -0.52
#define B  0.57

unsigned int window_width = 1024;
unsigned int window_height = 1024;

float xMin = -2.0;
float xMax =  2.0;
float yMin = -2.0;
float yMax =  2.0;

float steps = 1024.0;
float stepSizeX = (xMax - xMin)/steps;
float stepSizeY = (yMax - yMin)/steps;

float color (float x, float y) 
{
	float mag,maxMag,t1;
	float maxCount = 200;
	float count = 0;
	maxMag = 10;
	mag = 0.0;
	
	while (mag < maxMag && count < maxCount) 
	{
		t1 = x;	
		x = x*x - y*y + A;
		y = (2.0 * t1 * y) + B;
		mag = sqrt(x*x + y*y);
		count++;
	}
	if(count < maxCount) 
	{
		return(1.0);
	}
	else
	{
		return(0.0);
	}
}

void display(void) 
{ 
	float *pixels; 
	int k;

	pixels = (float *)malloc(window_width*window_height*3*sizeof(float));
	k=0;
	float x = xMin;
	float y = yMin;
	while(x <= xMax) 
	{
		y = yMin;
		while(y <= yMax) 
		{
			pixels[k] = color(x,y);
			pixels[k+1] = 0.0; 
			pixels[k+2] = 0.0;
			k=k+3;
			y += stepSizeY;
		}
		x += stepSizeX;
	}

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels); 
	glFlush(); 
}

int main(int argc, char** argv)
{ 
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	glutMainLoop();
}

